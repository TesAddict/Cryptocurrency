#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <time.h>
extern "C" { 
	#include "block_header.h"
	}

#define DEVICE 0

#define SHR(x,n) (x>>n)

#define ROTR(x,n) (SHR(x,n) | (x<<(32-n)))

#define s0(x) (ROTR(x,7)^ROTR(x,18)^SHR(x,3))
#define s1(x) (ROTR(x,17)^ROTR(x,19)^SHR(x,10))

#define S1(x) (ROTR(x,6)^ROTR(x,11)^ROTR(x,25))
#define S2(x) (ROTR(x,2)^ROTR(x,13)^ROTR(x,22))

#define F0(x, y, z) ((x & y) | (z & (x | y)))
#define F1(x, y, z) (z ^ (x & (y ^ z)))

#define P(a, b, c, d, e, f, g, h, x, K)      \
  {                                          \
    temp1 = h + S1(e) + F1(e, f, g) + K + x; \
    temp2 = S2(a) + F0(a, b, c);             \
    d += temp1;                              \
    h = temp1 + temp2;                       \
  }

__device__
static const uint32_t h[8] =
{
	0x6a09e667,
	0xbb67ae85,
	0x3c6ef372,
	0xa54ff53a,
	0x510e527f,
	0x9b05688c,
	0x1f83d9ab,
	0x5be0cd19
};

__device__
static const uint32_t K[64] =
{
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
  	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
   	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
   	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
   	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
   	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
   	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
   	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};


__device__ uint32_t h0[8];
__device__ uint32_t target[8];
__device__ bool computeH0 = false;

__device__
void computeTarget(uint32_t nbits)
{
	uint32_t mantissa = nbits & 0x00FFFFFF;
	uint32_t exponent = (nbits & 0xFF000000) >> 24;
	uint32_t offset = 0x20 - (exponent - 0x03);
	uint8_t inter[32]={0};
	
	inter[offset-1]   = (mantissa & 0x000000FF);

	inter[offset-2] = (mantissa & 0x0000FF00) >> 8;

	inter[offset-3] = (mantissa & 0x00FF0000) >> 16;

	target[0] = inter[0]<<24  | inter[1]<<16  | inter[2]<<8  | inter[3];
	target[1] = inter[4]<<24  | inter[5]<<16  | inter[6]<<8  | inter[7];
	target[2] = inter[8]<<24  | inter[9]<<16  | inter[10]<<8 | inter[11];
	target[3] = inter[12]<<24 | inter[13]<<16 | inter[14]<<8 | inter[15];
	target[4] = inter[16]<<24 | inter[17]<<16 | inter[18]<<8 | inter[19];
	target[5] = inter[20]<<24 | inter[21]<<16 | inter[22]<<8 | inter[23];
	target[6] = inter[24]<<24 | inter[25]<<16 | inter[26]<<8 | inter[27];
	target[7] = inter[28]<<24 | inter[29]<<16 | inter[30]<<8 | inter[31];
}

__global__
void sha256ComputeH2(uint32_t *block_header, uint32_t threads, uint32_t *h1, uint32_t *h2,unsigned long long int *counter)
{	
	
	uint32_t w[64];
	uint32_t A,B,C,D,E,F,G,H,temp1,temp2,nonce,timestamp;

 
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	long int idx_mod = idx;

	while(1)
	{
		if (computeH0 == false)
		{
			w[0]  = block_header[0];
			w[1]  = block_header[1];
			w[2]  = block_header[2];
			w[3]  = block_header[3];
			w[4]  = block_header[4];
			w[5]  = block_header[5];
			w[6]  = block_header[6];
			w[7]  = block_header[7];	
			w[8]  = block_header[8];
			w[9]  = block_header[9];
			w[10] = block_header[10];
			w[11] = block_header[11];
			w[12] = block_header[12];
			w[13] = block_header[13];
			w[14] = block_header[14];
			w[15] = block_header[15];
			w[16] = w[0]+(s0(w[1]))+w[9]+(s1(w[14]));
			w[17] = w[1]+(s0(w[2]))+w[10]+(s1(w[15]));
			w[18] = w[2]+(s0(w[3]))+w[11]+(s1(w[16]));
			w[19] = w[3]+(s0(w[4]))+w[12]+(s1(w[17]));
			w[20] = w[4]+(s0(w[5]))+w[13]+(s1(w[18]));
			w[21] = w[5]+(s0(w[6]))+w[14]+(s1(w[19]));
			w[22] = w[6]+(s0(w[7]))+w[15]+(s1(w[20]));
			w[23] = w[7]+(s0(w[8]))+w[16]+(s1(w[21]));
			w[24] = w[8]+(s0(w[9]))+w[17]+(s1(w[22]));
			w[25] = w[9]+(s0(w[10]))+w[18]+(s1(w[23]));
			w[26] = w[10]+(s0(w[11]))+w[19]+(s1(w[24]));
			w[27] = w[11]+(s0(w[12]))+w[20]+(s1(w[25]));
			w[28] = w[12]+(s0(w[13]))+w[21]+(s1(w[26]));
			w[29] = w[13]+(s0(w[14]))+w[22]+(s1(w[27]));
			w[30] = w[14]+(s0(w[15]))+w[23]+(s1(w[28]));
			w[31] = w[15]+(s0(w[16]))+w[24]+(s1(w[29]));
			w[32] = w[16]+(s0(w[17]))+w[25]+(s1(w[30]));
			w[33] = w[17]+(s0(w[18]))+w[26]+(s1(w[31]));
			w[34] = w[18]+(s0(w[19]))+w[27]+(s1(w[32]));
			w[35] = w[19]+(s0(w[20]))+w[28]+(s1(w[33]));
			w[36] = w[20]+(s0(w[21]))+w[29]+(s1(w[34]));
			w[37] = w[21]+(s0(w[22]))+w[30]+(s1(w[35]));
			w[38] = w[22]+(s0(w[23]))+w[31]+(s1(w[36]));
			w[39] = w[23]+(s0(w[24]))+w[32]+(s1(w[37]));
			w[40] = w[24]+(s0(w[25]))+w[33]+(s1(w[38]));
			w[41] = w[25]+(s0(w[26]))+w[34]+(s1(w[39]));
			w[42] = w[26]+(s0(w[27]))+w[35]+(s1(w[40]));
			w[43] = w[27]+(s0(w[28]))+w[36]+(s1(w[41]));
			w[44] = w[28]+(s0(w[29]))+w[37]+(s1(w[42]));
			w[45] = w[29]+(s0(w[30]))+w[38]+(s1(w[43]));
			w[46] = w[30]+(s0(w[31]))+w[39]+(s1(w[44]));
			w[47] = w[31]+(s0(w[32]))+w[40]+(s1(w[45]));
			w[48] = w[32]+(s0(w[33]))+w[41]+(s1(w[46]));
			w[49] = w[33]+(s0(w[34]))+w[42]+(s1(w[47]));
			w[50] = w[34]+(s0(w[35]))+w[43]+(s1(w[48]));
			w[51] = w[35]+(s0(w[36]))+w[44]+(s1(w[49]));
			w[52] = w[36]+(s0(w[37]))+w[45]+(s1(w[50]));
			w[53] = w[37]+(s0(w[38]))+w[46]+(s1(w[51]));
			w[54] = w[38]+(s0(w[39]))+w[47]+(s1(w[52]));
			w[55] = w[39]+(s0(w[40]))+w[48]+(s1(w[53]));
			w[56] = w[40]+(s0(w[41]))+w[49]+(s1(w[54]));
			w[57] = w[41]+(s0(w[42]))+w[50]+(s1(w[55]));
			w[58] = w[42]+(s0(w[43]))+w[51]+(s1(w[56]));
			w[59] = w[43]+(s0(w[44]))+w[52]+(s1(w[57]));
			w[60] = w[44]+(s0(w[45]))+w[53]+(s1(w[58]));
			w[61] = w[45]+(s0(w[46]))+w[54]+(s1(w[59]));
			w[62] = w[46]+(s0(w[47]))+w[55]+(s1(w[60]));
			w[63] = w[47]+(s0(w[48]))+w[56]+(s1(w[61]));


			A = h[0];
			B = h[1];
			C = h[2];
			D = h[3];
			E = h[4];
			F = h[5];
			G = h[6];
			H = h[7];

	
			P(A, B, C, D, E, F, G, H, w[0], K[0]);  
			P(H, A, B, C, D, E, F, G, w[1], K[1]); 
			P(G, H, A, B, C, D, E, F, w[2], K[2]);  
			P(F, G, H, A, B, C, D, E, w[3], K[3]);  
			P(E, F, G, H, A, B, C, D, w[4], K[4]);
			P(D, E, F, G, H, A, B, C, w[5], K[5]);
			P(C, D, E, F, G, H, A, B, w[6], K[6]);
			P(B, C, D, E, F, G, H, A, w[7], K[7]);

			P(A, B, C, D, E, F, G, H, w[8], K[8]);  
			P(H, A, B, C, D, E, F, G, w[9], K[9]); 
			P(G, H, A, B, C, D, E, F, w[10], K[10]);  
			P(F, G, H, A, B, C, D, E, w[11], K[11]);  
			P(E, F, G, H, A, B, C, D, w[12], K[12]);
			P(D, E, F, G, H, A, B, C, w[13], K[13]);
			P(C, D, E, F, G, H, A, B, w[14], K[14]);
			P(B, C, D, E, F, G, H, A, w[15], K[15]);

			P(A, B, C, D, E, F, G, H, w[16], K[16]);  
			P(H, A, B, C, D, E, F, G, w[17], K[17]); 
			P(G, H, A, B, C, D, E, F, w[18], K[18]);  
			P(F, G, H, A, B, C, D, E, w[19], K[19]);  
			P(E, F, G, H, A, B, C, D, w[20], K[20]);
			P(D, E, F, G, H, A, B, C, w[21], K[21]);
			P(C, D, E, F, G, H, A, B, w[22], K[22]);
			P(B, C, D, E, F, G, H, A, w[23], K[23]);

			P(A, B, C, D, E, F, G, H, w[24], K[24]);  
			P(H, A, B, C, D, E, F, G, w[25], K[25]); 
			P(G, H, A, B, C, D, E, F, w[26], K[26]);  
			P(F, G, H, A, B, C, D, E, w[27], K[27]);  
			P(E, F, G, H, A, B, C, D, w[28], K[28]);
			P(D, E, F, G, H, A, B, C, w[29], K[29]);
			P(C, D, E, F, G, H, A, B, w[30], K[30]);
			P(B, C, D, E, F, G, H, A, w[31], K[31]);

			P(A, B, C, D, E, F, G, H, w[32], K[32]);  
			P(H, A, B, C, D, E, F, G, w[33], K[33]); 
			P(G, H, A, B, C, D, E, F, w[34], K[34]);  
			P(F, G, H, A, B, C, D, E, w[35], K[35]);  
			P(E, F, G, H, A, B, C, D, w[36], K[36]);
			P(D, E, F, G, H, A, B, C, w[37], K[37]);
			P(C, D, E, F, G, H, A, B, w[38], K[38]);
			P(B, C, D, E, F, G, H, A, w[39], K[39]);

			P(A, B, C, D, E, F, G, H, w[40], K[40]);  
			P(H, A, B, C, D, E, F, G, w[41], K[41]); 
			P(G, H, A, B, C, D, E, F, w[42], K[42]);  
			P(F, G, H, A, B, C, D, E, w[43], K[43]);  
			P(E, F, G, H, A, B, C, D, w[44], K[44]);
			P(D, E, F, G, H, A, B, C, w[45], K[45]);
			P(C, D, E, F, G, H, A, B, w[46], K[46]);
			P(B, C, D, E, F, G, H, A, w[47], K[47]);

			P(A, B, C, D, E, F, G, H, w[48], K[48]);  
			P(H, A, B, C, D, E, F, G, w[49], K[49]); 
			P(G, H, A, B, C, D, E, F, w[50], K[50]);  
			P(F, G, H, A, B, C, D, E, w[51], K[51]);  
			P(E, F, G, H, A, B, C, D, w[52], K[52]);
			P(D, E, F, G, H, A, B, C, w[53], K[53]);
			P(C, D, E, F, G, H, A, B, w[54], K[54]);
			P(B, C, D, E, F, G, H, A, w[55], K[55]);

			P(A, B, C, D, E, F, G, H, w[56], K[56]);  
			P(H, A, B, C, D, E, F, G, w[57], K[57]); 
			P(G, H, A, B, C, D, E, F, w[58], K[58]);  
			P(F, G, H, A, B, C, D, E, w[59], K[59]);  
			P(E, F, G, H, A, B, C, D, w[60], K[60]);
			P(D, E, F, G, H, A, B, C, w[61], K[61]);
			P(C, D, E, F, G, H, A, B, w[62], K[62]);
			P(B, C, D, E, F, G, H, A, w[63], K[63]);

			if (idx == 0)
			{
				h0[0] = h[0]+A;
				h0[1] = h[1]+B;
				h0[2] = h[2]+C;
				h0[3] = h[3]+D;
				h0[4] = h[4]+E;
				h0[5] = h[5]+F;
				h0[6] = h[6]+G;
				h0[7] = h[7]+H;
				computeTarget(block_header[18]);
				computeH0 = true;
			}
			__syncthreads();
		}

		
		w[0]  = block_header[16];
		w[1]  = block_header[17];
		w[2]  = block_header[18];
		w[3]  = block_header[19]+idx_mod;
		nonce = w[3];
		w[4]  = 0x80000000; 
		w[5]  = 0x00000000;
		w[6]  = 0x00000000;
		w[7]  = 0x00000000;	
		w[8]  = 0x00000000;
		w[9]  = 0x00000000;
		w[10] = 0x00000000;
		w[11] = 0x00000000;
		w[12] = 0x00000000;
		w[13] = 0x00000000;
		w[14] = 0x00000000;
		w[15] = 0x00000280;
		w[16] = w[0]+(s0(w[1]))+w[9]+(s1(w[14]));
		w[17] = w[1]+(s0(w[2]))+w[10]+(s1(w[15]));
		w[18] = w[2]+(s0(w[3]))+w[11]+(s1(w[16]));
		w[19] = w[3]+(s0(w[4]))+w[12]+(s1(w[17]));
		w[20] = w[4]+(s0(w[5]))+w[13]+(s1(w[18]));
		w[21] = w[5]+(s0(w[6]))+w[14]+(s1(w[19]));
		w[22] = w[6]+(s0(w[7]))+w[15]+(s1(w[20]));
		w[23] = w[7]+(s0(w[8]))+w[16]+(s1(w[21]));
		w[24] = w[8]+(s0(w[9]))+w[17]+(s1(w[22]));
		w[25] = w[9]+(s0(w[10]))+w[18]+(s1(w[23]));
		w[26] = w[10]+(s0(w[11]))+w[19]+(s1(w[24]));
		w[27] = w[11]+(s0(w[12]))+w[20]+(s1(w[25]));
		w[28] = w[12]+(s0(w[13]))+w[21]+(s1(w[26]));
		w[29] = w[13]+(s0(w[14]))+w[22]+(s1(w[27]));
		w[30] = w[14]+(s0(w[15]))+w[23]+(s1(w[28]));
		w[31] = w[15]+(s0(w[16]))+w[24]+(s1(w[29]));
		w[32] = w[16]+(s0(w[17]))+w[25]+(s1(w[30]));
		w[33] = w[17]+(s0(w[18]))+w[26]+(s1(w[31]));
		w[34] = w[18]+(s0(w[19]))+w[27]+(s1(w[32]));
		w[35] = w[19]+(s0(w[20]))+w[28]+(s1(w[33]));
		w[36] = w[20]+(s0(w[21]))+w[29]+(s1(w[34]));
		w[37] = w[21]+(s0(w[22]))+w[30]+(s1(w[35]));
		w[38] = w[22]+(s0(w[23]))+w[31]+(s1(w[36]));
		w[39] = w[23]+(s0(w[24]))+w[32]+(s1(w[37]));
		w[40] = w[24]+(s0(w[25]))+w[33]+(s1(w[38]));
		w[41] = w[25]+(s0(w[26]))+w[34]+(s1(w[39]));
		w[42] = w[26]+(s0(w[27]))+w[35]+(s1(w[40]));
		w[43] = w[27]+(s0(w[28]))+w[36]+(s1(w[41]));
		w[44] = w[28]+(s0(w[29]))+w[37]+(s1(w[42]));
		w[45] = w[29]+(s0(w[30]))+w[38]+(s1(w[43]));
		w[46] = w[30]+(s0(w[31]))+w[39]+(s1(w[44]));
		w[47] = w[31]+(s0(w[32]))+w[40]+(s1(w[45]));
		w[48] = w[32]+(s0(w[33]))+w[41]+(s1(w[46]));
		w[49] = w[33]+(s0(w[34]))+w[42]+(s1(w[47]));
		w[50] = w[34]+(s0(w[35]))+w[43]+(s1(w[48]));
		w[51] = w[35]+(s0(w[36]))+w[44]+(s1(w[49]));
		w[52] = w[36]+(s0(w[37]))+w[45]+(s1(w[50]));
		w[53] = w[37]+(s0(w[38]))+w[46]+(s1(w[51]));
		w[54] = w[38]+(s0(w[39]))+w[47]+(s1(w[52]));
		w[55] = w[39]+(s0(w[40]))+w[48]+(s1(w[53]));
		w[56] = w[40]+(s0(w[41]))+w[49]+(s1(w[54]));
		w[57] = w[41]+(s0(w[42]))+w[50]+(s1(w[55]));
		w[58] = w[42]+(s0(w[43]))+w[51]+(s1(w[56]));
		w[59] = w[43]+(s0(w[44]))+w[52]+(s1(w[57]));
		w[60] = w[44]+(s0(w[45]))+w[53]+(s1(w[58]));
		w[61] = w[45]+(s0(w[46]))+w[54]+(s1(w[59]));
		w[62] = w[46]+(s0(w[47]))+w[55]+(s1(w[60]));
		w[63] = w[47]+(s0(w[48]))+w[56]+(s1(w[61]));

		A = h0[0];
		B = h0[1];
		C = h0[2];
		D = h0[3];
		E = h0[4];
		F = h0[5];
		G = h0[6];
		H = h0[7];

		P(A, B, C, D, E, F, G, H, w[0], K[0]);  
		P(H, A, B, C, D, E, F, G, w[1], K[1]); 
		P(G, H, A, B, C, D, E, F, w[2], K[2]);  
		P(F, G, H, A, B, C, D, E, w[3], K[3]);  
		P(E, F, G, H, A, B, C, D, w[4], K[4]);
		P(D, E, F, G, H, A, B, C, w[5], K[5]);
		P(C, D, E, F, G, H, A, B, w[6], K[6]);
		P(B, C, D, E, F, G, H, A, w[7], K[7]);

		P(A, B, C, D, E, F, G, H, w[8], K[8]);  
		P(H, A, B, C, D, E, F, G, w[9], K[9]); 
		P(G, H, A, B, C, D, E, F, w[10], K[10]);  
		P(F, G, H, A, B, C, D, E, w[11], K[11]);  
		P(E, F, G, H, A, B, C, D, w[12], K[12]);
		P(D, E, F, G, H, A, B, C, w[13], K[13]);
		P(C, D, E, F, G, H, A, B, w[14], K[14]);
		P(B, C, D, E, F, G, H, A, w[15], K[15]);

		P(A, B, C, D, E, F, G, H, w[16], K[16]);  
		P(H, A, B, C, D, E, F, G, w[17], K[17]); 
		P(G, H, A, B, C, D, E, F, w[18], K[18]);  
		P(F, G, H, A, B, C, D, E, w[19], K[19]);  
		P(E, F, G, H, A, B, C, D, w[20], K[20]);
		P(D, E, F, G, H, A, B, C, w[21], K[21]);
		P(C, D, E, F, G, H, A, B, w[22], K[22]);
		P(B, C, D, E, F, G, H, A, w[23], K[23]);

		P(A, B, C, D, E, F, G, H, w[24], K[24]);  
		P(H, A, B, C, D, E, F, G, w[25], K[25]); 
		P(G, H, A, B, C, D, E, F, w[26], K[26]);  
		P(F, G, H, A, B, C, D, E, w[27], K[27]);  
		P(E, F, G, H, A, B, C, D, w[28], K[28]);
		P(D, E, F, G, H, A, B, C, w[29], K[29]);
		P(C, D, E, F, G, H, A, B, w[30], K[30]);
		P(B, C, D, E, F, G, H, A, w[31], K[31]);

		P(A, B, C, D, E, F, G, H, w[32], K[32]);  
		P(H, A, B, C, D, E, F, G, w[33], K[33]); 
		P(G, H, A, B, C, D, E, F, w[34], K[34]);  
		P(F, G, H, A, B, C, D, E, w[35], K[35]);  
		P(E, F, G, H, A, B, C, D, w[36], K[36]);
		P(D, E, F, G, H, A, B, C, w[37], K[37]);
		P(C, D, E, F, G, H, A, B, w[38], K[38]);
		P(B, C, D, E, F, G, H, A, w[39], K[39]);

		P(A, B, C, D, E, F, G, H, w[40], K[40]);  
		P(H, A, B, C, D, E, F, G, w[41], K[41]); 
		P(G, H, A, B, C, D, E, F, w[42], K[42]);  
		P(F, G, H, A, B, C, D, E, w[43], K[43]);  
		P(E, F, G, H, A, B, C, D, w[44], K[44]);
		P(D, E, F, G, H, A, B, C, w[45], K[45]);
		P(C, D, E, F, G, H, A, B, w[46], K[46]);
		P(B, C, D, E, F, G, H, A, w[47], K[47]);

		P(A, B, C, D, E, F, G, H, w[48], K[48]);  
		P(H, A, B, C, D, E, F, G, w[49], K[49]); 
		P(G, H, A, B, C, D, E, F, w[50], K[50]);  
		P(F, G, H, A, B, C, D, E, w[51], K[51]);  
		P(E, F, G, H, A, B, C, D, w[52], K[52]);
		P(D, E, F, G, H, A, B, C, w[53], K[53]);
		P(C, D, E, F, G, H, A, B, w[54], K[54]);
		P(B, C, D, E, F, G, H, A, w[55], K[55]);

		P(A, B, C, D, E, F, G, H, w[56], K[56]);  
		P(H, A, B, C, D, E, F, G, w[57], K[57]); 
		P(G, H, A, B, C, D, E, F, w[58], K[58]);  
		P(F, G, H, A, B, C, D, E, w[59], K[59]);  
		P(E, F, G, H, A, B, C, D, w[60], K[60]);
		P(D, E, F, G, H, A, B, C, w[61], K[61]);
		P(C, D, E, F, G, H, A, B, w[62], K[62]);
		P(B, C, D, E, F, G, H, A, w[63], K[63]);


		w[0] = h0[0]+A;
		w[1] = h0[1]+B;
		w[2] = h0[2]+C;
		w[3] = h0[3]+D;
		w[4] = h0[4]+E;
		w[5] = h0[5]+F;
		w[6] = h0[6]+G;
		w[7] = h0[7]+H;
		w[8]  = 0x80000000;
		w[9]  = 0x00000000;
		w[10] = 0x00000000;
		w[11] = 0x00000000;
		w[12] = 0x00000000;
		w[13] = 0x00000000;
		w[14] = 0x00000000;
		w[15] = 0x00000100;
		w[16] = w[0]+(s0(w[1]))+w[9]+(s1(w[14]));
		w[17] = w[1]+(s0(w[2]))+w[10]+(s1(w[15]));
		w[18] = w[2]+(s0(w[3]))+w[11]+(s1(w[16]));
		w[19] = w[3]+(s0(w[4]))+w[12]+(s1(w[17]));
		w[20] = w[4]+(s0(w[5]))+w[13]+(s1(w[18]));
		w[21] = w[5]+(s0(w[6]))+w[14]+(s1(w[19]));
		w[22] = w[6]+(s0(w[7]))+w[15]+(s1(w[20]));
		w[23] = w[7]+(s0(w[8]))+w[16]+(s1(w[21]));
		w[24] = w[8]+(s0(w[9]))+w[17]+(s1(w[22]));
		w[25] = w[9]+(s0(w[10]))+w[18]+(s1(w[23]));
		w[26] = w[10]+(s0(w[11]))+w[19]+(s1(w[24]));
		w[27] = w[11]+(s0(w[12]))+w[20]+(s1(w[25]));
		w[28] = w[12]+(s0(w[13]))+w[21]+(s1(w[26]));
		w[29] = w[13]+(s0(w[14]))+w[22]+(s1(w[27]));
		w[30] = w[14]+(s0(w[15]))+w[23]+(s1(w[28]));
		w[31] = w[15]+(s0(w[16]))+w[24]+(s1(w[29]));
		w[32] = w[16]+(s0(w[17]))+w[25]+(s1(w[30]));
		w[33] = w[17]+(s0(w[18]))+w[26]+(s1(w[31]));
		w[34] = w[18]+(s0(w[19]))+w[27]+(s1(w[32]));
		w[35] = w[19]+(s0(w[20]))+w[28]+(s1(w[33]));
		w[36] = w[20]+(s0(w[21]))+w[29]+(s1(w[34]));
		w[37] = w[21]+(s0(w[22]))+w[30]+(s1(w[35]));
		w[38] = w[22]+(s0(w[23]))+w[31]+(s1(w[36]));
		w[39] = w[23]+(s0(w[24]))+w[32]+(s1(w[37]));
		w[40] = w[24]+(s0(w[25]))+w[33]+(s1(w[38]));
		w[41] = w[25]+(s0(w[26]))+w[34]+(s1(w[39]));
		w[42] = w[26]+(s0(w[27]))+w[35]+(s1(w[40]));
		w[43] = w[27]+(s0(w[28]))+w[36]+(s1(w[41]));
		w[44] = w[28]+(s0(w[29]))+w[37]+(s1(w[42]));
		w[45] = w[29]+(s0(w[30]))+w[38]+(s1(w[43]));
		w[46] = w[30]+(s0(w[31]))+w[39]+(s1(w[44]));
		w[47] = w[31]+(s0(w[32]))+w[40]+(s1(w[45]));
		w[48] = w[32]+(s0(w[33]))+w[41]+(s1(w[46]));
		w[49] = w[33]+(s0(w[34]))+w[42]+(s1(w[47]));
		w[50] = w[34]+(s0(w[35]))+w[43]+(s1(w[48]));
		w[51] = w[35]+(s0(w[36]))+w[44]+(s1(w[49]));
		w[52] = w[36]+(s0(w[37]))+w[45]+(s1(w[50]));
		w[53] = w[37]+(s0(w[38]))+w[46]+(s1(w[51]));
		w[54] = w[38]+(s0(w[39]))+w[47]+(s1(w[52]));
		w[55] = w[39]+(s0(w[40]))+w[48]+(s1(w[53]));
		w[56] = w[40]+(s0(w[41]))+w[49]+(s1(w[54]));
		w[57] = w[41]+(s0(w[42]))+w[50]+(s1(w[55]));
		w[58] = w[42]+(s0(w[43]))+w[51]+(s1(w[56]));
		w[59] = w[43]+(s0(w[44]))+w[52]+(s1(w[57]));
		w[60] = w[44]+(s0(w[45]))+w[53]+(s1(w[58]));
		w[61] = w[45]+(s0(w[46]))+w[54]+(s1(w[59]));
		w[62] = w[46]+(s0(w[47]))+w[55]+(s1(w[60]));
		w[63] = w[47]+(s0(w[48]))+w[56]+(s1(w[61]));

		A = h[0];
		B = h[1];
		C = h[2];
		D = h[3];
		E = h[4];
		F = h[5];
		G = h[6];
		H = h[7];

		P(A, B, C, D, E, F, G, H, w[0], K[0]);  
		P(H, A, B, C, D, E, F, G, w[1], K[1]); 
		P(G, H, A, B, C, D, E, F, w[2], K[2]);  
		P(F, G, H, A, B, C, D, E, w[3], K[3]);  
		P(E, F, G, H, A, B, C, D, w[4], K[4]);
		P(D, E, F, G, H, A, B, C, w[5], K[5]);
		P(C, D, E, F, G, H, A, B, w[6], K[6]);
		P(B, C, D, E, F, G, H, A, w[7], K[7]);

		P(A, B, C, D, E, F, G, H, w[8], K[8]);  
		P(H, A, B, C, D, E, F, G, w[9], K[9]); 
		P(G, H, A, B, C, D, E, F, w[10], K[10]);  
		P(F, G, H, A, B, C, D, E, w[11], K[11]);  
		P(E, F, G, H, A, B, C, D, w[12], K[12]);
		P(D, E, F, G, H, A, B, C, w[13], K[13]);
		P(C, D, E, F, G, H, A, B, w[14], K[14]);
		P(B, C, D, E, F, G, H, A, w[15], K[15]);

		P(A, B, C, D, E, F, G, H, w[16], K[16]);  
		P(H, A, B, C, D, E, F, G, w[17], K[17]); 
		P(G, H, A, B, C, D, E, F, w[18], K[18]);  
		P(F, G, H, A, B, C, D, E, w[19], K[19]);  
		P(E, F, G, H, A, B, C, D, w[20], K[20]);
		P(D, E, F, G, H, A, B, C, w[21], K[21]);
		P(C, D, E, F, G, H, A, B, w[22], K[22]);
		P(B, C, D, E, F, G, H, A, w[23], K[23]);

		P(A, B, C, D, E, F, G, H, w[24], K[24]);  
		P(H, A, B, C, D, E, F, G, w[25], K[25]); 
		P(G, H, A, B, C, D, E, F, w[26], K[26]);  
		P(F, G, H, A, B, C, D, E, w[27], K[27]);  
		P(E, F, G, H, A, B, C, D, w[28], K[28]);
		P(D, E, F, G, H, A, B, C, w[29], K[29]);
		P(C, D, E, F, G, H, A, B, w[30], K[30]);
		P(B, C, D, E, F, G, H, A, w[31], K[31]);

		P(A, B, C, D, E, F, G, H, w[32], K[32]);  
		P(H, A, B, C, D, E, F, G, w[33], K[33]); 
		P(G, H, A, B, C, D, E, F, w[34], K[34]);  
		P(F, G, H, A, B, C, D, E, w[35], K[35]);  
		P(E, F, G, H, A, B, C, D, w[36], K[36]);
		P(D, E, F, G, H, A, B, C, w[37], K[37]);
		P(C, D, E, F, G, H, A, B, w[38], K[38]);
		P(B, C, D, E, F, G, H, A, w[39], K[39]);

		P(A, B, C, D, E, F, G, H, w[40], K[40]);  
		P(H, A, B, C, D, E, F, G, w[41], K[41]); 
		P(G, H, A, B, C, D, E, F, w[42], K[42]);  
		P(F, G, H, A, B, C, D, E, w[43], K[43]);  
		P(E, F, G, H, A, B, C, D, w[44], K[44]);
		P(D, E, F, G, H, A, B, C, w[45], K[45]);
		P(C, D, E, F, G, H, A, B, w[46], K[46]);
		P(B, C, D, E, F, G, H, A, w[47], K[47]);

		P(A, B, C, D, E, F, G, H, w[48], K[48]);  
		P(H, A, B, C, D, E, F, G, w[49], K[49]); 
		P(G, H, A, B, C, D, E, F, w[50], K[50]);  
		P(F, G, H, A, B, C, D, E, w[51], K[51]);  
		P(E, F, G, H, A, B, C, D, w[52], K[52]);
		P(D, E, F, G, H, A, B, C, w[53], K[53]);
		P(C, D, E, F, G, H, A, B, w[54], K[54]);
		P(B, C, D, E, F, G, H, A, w[55], K[55]);

		P(A, B, C, D, E, F, G, H, w[56], K[56]);  
		P(H, A, B, C, D, E, F, G, w[57], K[57]); 
		P(G, H, A, B, C, D, E, F, w[58], K[58]);  
		P(F, G, H, A, B, C, D, E, w[59], K[59]);  
		P(E, F, G, H, A, B, C, D, w[60], K[60]);
		P(D, E, F, G, H, A, B, C, w[61], K[61]);
		P(C, D, E, F, G, H, A, B, w[62], K[62]);
		P(B, C, D, E, F, G, H, A, w[63], K[63]);

		
		if (h[0]+A<=target[0])
		{
			atomicExch(&h1[0],w[0]);
			atomicExch(&h1[1],w[1]);
			atomicExch(&h1[2],w[2]);
			atomicExch(&h1[3],w[3]);
			atomicExch(&h1[4],w[4]);
			atomicExch(&h1[5],w[5]);
			atomicExch(&h1[6],w[6]);
			atomicExch(&h1[7],w[7]);

			atomicExch(&h2[0],h[0]+A);
			atomicExch(&h2[1],h[1]+B);
			atomicExch(&h2[2],h[2]+C);
			atomicExch(&h2[3],h[3]+D);
			atomicExch(&h2[4],h[4]+E);
			atomicExch(&h2[5],h[5]+F);
			atomicExch(&h2[6],h[6]+G);
			atomicExch(&h2[7],h[7]+H);
			
			atomicExch(&block_header[19],nonce);
			asm("trap;");
		}	
		idx_mod += threads;

		/* 
		This is problematic. The current check for overflow leads to
		incorrect results. 
		*/
		//if (idx_mod >= 0xffffffff)
		//{
		//	idx_mod = idx;
		//	computeH0 = false;
		//}
		atomicAdd(counter,1);
		__syncthreads();
	}
}


int main(int argc, char *argv[])
{
	int threads_per_block;
	int blocks;
	char *filename;
	
	
	threads_per_block = atoi(argv[1]);
	blocks = atoi(argv[2]);
	filename = argv[3];
	
	int device;
	hipGetDeviceCount(&device);

	printf("%d cuda device(s)\n", device);

	hipDeviceProp_t prop;
	for (int dev = 0; dev < device; dev++) 
	{
		hipGetDeviceProperties(&prop, dev);
		printf("%s compute capability ", prop.name);
		printf("%d.%d\n", prop.major, prop.minor);
	}
	
	device = DEVICE;
	hipGetDeviceProperties(&prop, device);
	printf("Using %s\n\n", prop.name);
	hipSetDevice(device);

	int threads = threads_per_block * blocks;

	uint32_t *block_header;
	uint32_t *h1;
	uint32_t *h2;
	unsigned long long int *counter;

	hipMallocManaged(&block_header, 640);
	hipMallocManaged(&h1, 256);
	hipMallocManaged(&h2, 256);
	hipMallocManaged(&counter, 64);


	getBlockHeader(block_header,filename);

	block_header[17] = time(NULL);
	
	clock_t start, end;
	start = clock();

	sha256ComputeH2<<<blocks, threads_per_block>>>(block_header,threads,h1,h2,counter);
	hipDeviceSynchronize();

	for(int i=0;i<20;i++)
		printf("%08x", block_header[i]);
	printf("\n");

	for(int i=0;i<8;i++)
		printf("%08x", h1[i]);
	printf("\n");

	for(int i=0;i<8;i++)
		printf("%08x", h2[i]);
	printf("\n");

	end = clock();
	double cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

	printf("%lld hashes attempted in %f seconds.\n", counter[0],cpu_time_used);
	printf("%f MH/s\n", counter[0]/cpu_time_used/1000000);
	
	hipFree(block_header);
	hipFree(h1);
	hipFree(h2);
	hipFree(counter);
	hipDeviceReset();

	return 0;
}
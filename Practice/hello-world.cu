
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
	printf("Hello World from GPU thread: %d\n", threadIdx.x);
}

int main(void)
{
	printf("Hello World from the CPU!\n");
	
	helloFromGPU<<<1, 10>>>();
	hipDeviceReset();
	return 0;
}
#include "hip/hip_runtime.h"
/*
Author: Eleftherios Amperiadis
Date: 02.17.2019
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <stddef.h>
#include <string.h>
#include <stdbool.h>
#include <time.h>
#include "sha512_init.c"

void verifyLeadingZeroes(unsigned char *hash, int leading_zero, int hash_offset, bool* hashed_winner,
	int idx);

#define UL64(x) x##ULL

#define SHR(x, n) (x >> n)
#define ROTR(x, n) (SHR(x, n) | (x << (64 - n)))

#define S0(x) (ROTR(x, 1) ^ ROTR(x, 8) ^ SHR(x, 7))
#define S1(x) (ROTR(x, 19) ^ ROTR(x, 61) ^ SHR(x, 6))

#define S2(x) (ROTR(x, 28) ^ ROTR(x, 34) ^ ROTR(x, 39))
#define S3(x) (ROTR(x, 14) ^ ROTR(x, 18) ^ ROTR(x, 41))

#define F0(x, y, z) ((x & y) | (z & (x | y)))
#define F1(x, y, z) (z ^ (x & (y ^ z)))

#define P(a, b, c, d, e, f, g, h, x, K)      \
  {                                          \
    temp1 = h + S3(e) + F1(e, f, g) + K + x; \
    temp2 = S2(a) + F0(a, b, c);             \
    d += temp1;                              \
    h = temp1 + temp2;                       \
  }


__device__ static const uint64_t K[80] = 
{
    UL64(0x428A2F98D728AE22), UL64(0x7137449123EF65CD),
    UL64(0xB5C0FBCFEC4D3B2F), UL64(0xE9B5DBA58189DBBC),
    UL64(0x3956C25BF348B538), UL64(0x59F111F1B605D019),
    UL64(0x923F82A4AF194F9B), UL64(0xAB1C5ED5DA6D8118),
    UL64(0xD807AA98A3030242), UL64(0x12835B0145706FBE),
    UL64(0x243185BE4EE4B28C), UL64(0x550C7DC3D5FFB4E2),
    UL64(0x72BE5D74F27B896F), UL64(0x80DEB1FE3B1696B1),
    UL64(0x9BDC06A725C71235), UL64(0xC19BF174CF692694),
    UL64(0xE49B69C19EF14AD2), UL64(0xEFBE4786384F25E3),
    UL64(0x0FC19DC68B8CD5B5), UL64(0x240CA1CC77AC9C65),
    UL64(0x2DE92C6F592B0275), UL64(0x4A7484AA6EA6E483),
    UL64(0x5CB0A9DCBD41FBD4), UL64(0x76F988DA831153B5),
    UL64(0x983E5152EE66DFAB), UL64(0xA831C66D2DB43210),
    UL64(0xB00327C898FB213F), UL64(0xBF597FC7BEEF0EE4),
    UL64(0xC6E00BF33DA88FC2), UL64(0xD5A79147930AA725),
    UL64(0x06CA6351E003826F), UL64(0x142929670A0E6E70),
    UL64(0x27B70A8546D22FFC), UL64(0x2E1B21385C26C926),
    UL64(0x4D2C6DFC5AC42AED), UL64(0x53380D139D95B3DF),
    UL64(0x650A73548BAF63DE), UL64(0x766A0ABB3C77B2A8),
    UL64(0x81C2C92E47EDAEE6), UL64(0x92722C851482353B),
    UL64(0xA2BFE8A14CF10364), UL64(0xA81A664BBC423001),
    UL64(0xC24B8B70D0F89791), UL64(0xC76C51A30654BE30),
    UL64(0xD192E819D6EF5218), UL64(0xD69906245565A910),
    UL64(0xF40E35855771202A), UL64(0x106AA07032BBD1B8),
    UL64(0x19A4C116B8D2D0C8), UL64(0x1E376C085141AB53),
    UL64(0x2748774CDF8EEB99), UL64(0x34B0BCB5E19B48A8),
    UL64(0x391C0CB3C5C95A63), UL64(0x4ED8AA4AE3418ACB),
    UL64(0x5B9CCA4F7763E373), UL64(0x682E6FF3D6B2B8A3),
    UL64(0x748F82EE5DEFB2FC), UL64(0x78A5636F43172F60),
    UL64(0x84C87814A1F0AB72), UL64(0x8CC702081A6439EC),
    UL64(0x90BEFFFA23631E28), UL64(0xA4506CEBDE82BDE9),
    UL64(0xBEF9A3F7B2C67915), UL64(0xC67178F2E372532B),
    UL64(0xCA273ECEEA26619C), UL64(0xD186B8C721C0C207),
    UL64(0xEADA7DD6CDE0EB1E), UL64(0xF57D4F7FEE6ED178),
    UL64(0x06F067AA72176FBA), UL64(0x0A637DC5A2C898A6),
    UL64(0x113F9804BEF90DAE), UL64(0x1B710B35131C471B),
    UL64(0x28DB77F523047D84), UL64(0x32CAAB7B40C72493),
    UL64(0x3C9EBE0A15C9BEBC), UL64(0x431D67C49C100D4C),
    UL64(0x4CC5D4BECB3E42B6), UL64(0x597F299CFC657E2A),
    UL64(0x5FCB6FAB3AD6FAEC), UL64(0x6C44198C4A475817)
};

__device__ static const uint64_t H_array[8] = 
{
   	UL64(0x6A09E667F3BCC908),
	UL64(0xBB67AE8584CAA73B),
	UL64(0x3C6EF372FE94F82B),
	UL64(0xA54FF53A5F1D36F1),
	UL64(0x510E527FADE682D1),
	UL64(0x9B05688C2B3E6C1F),
	UL64(0x1F83D9ABFB41BD6B),
	UL64(0x5BE0CD19137E2179)
};

__device__
void computeHash(unsigned char *padded_array, int size, unsigned char *hashed_array,
	int idx, bool* hashed_winner)
{
	uint64_t s0, s1;
	uint64_t w[80];
	uint64_t A, B, C, D, E, F, G, H, temp1, temp2;
	uint64_t state[8];

	int thread_offset_pad = idx*128;
	int thread_offset_sha512 = idx*64;

	for(int i=0;i<16;i++)
	{	
		for(int j=0;j<8;j++)
		{
			w[i] <<= 8;
			w[i] |= (uint64_t)padded_array[i*8+j+thread_offset_pad];
		}
	}

	for(int i=16;i<80;i++)
	{	
			s0 = S0(w[i-15]);
			s1 = S1(w[i-2]);
			w[i] = w[i-16] + s0 + w[i-7] + s1;		
	}

	A = H_array[0];
  	B = H_array[1];
  	C = H_array[2];
  	D = H_array[3];
  	E = H_array[4];
  	F = H_array[5];
  	G = H_array[6];
  	H = H_array[7];
  	int i = 0;

  	do {
    P(A, B, C, D, E, F, G, H, w[i], K[i]);
    i++;
    P(H, A, B, C, D, E, F, G, w[i], K[i]);
    i++;
    P(G, H, A, B, C, D, E, F, w[i], K[i]);
    i++;
    P(F, G, H, A, B, C, D, E, w[i], K[i]);
    i++;
    P(E, F, G, H, A, B, C, D, w[i], K[i]);
    i++;
    P(D, E, F, G, H, A, B, C, w[i], K[i]);
    i++;
    P(C, D, E, F, G, H, A, B, w[i], K[i]);
    i++;
    P(B, C, D, E, F, G, H, A, w[i], K[i]);
    i++;
  	} while (i < 80);

  	state[0] = H_array[0];
  	state[1] = H_array[1];
  	state[2] = H_array[2];
  	state[3] = H_array[3];
  	state[4] = H_array[4];
  	state[5] = H_array[5];
  	state[6] = H_array[6];
  	state[7] = H_array[7];

  	state[0] += A;
  	state[1] += B;
  	state[2] += C;
  	state[3] += D;
  	state[4] += E;
  	state[5] += F;
  	state[6] += G;
  	state[7] += H;
	
  	for(int i=0;i<8;i++)
  	{
	  	hashed_array[thread_offset_sha512+(i*8)]    = state[i] >> 56;
		hashed_array[thread_offset_sha512+(i*8)+1]  = state[i] >> 48;
		hashed_array[thread_offset_sha512+(i*8)+2]  = state[i] >> 40;
		hashed_array[thread_offset_sha512+(i*8)+3]  = state[i] >> 32;
		hashed_array[thread_offset_sha512+(i*8)+4]  = state[i] >> 24;
	    hashed_array[thread_offset_sha512+(i*8)+5]  = state[i] >> 16;
	    hashed_array[thread_offset_sha512+(i*8)+6]  = state[i] >>  8;
	    hashed_array[thread_offset_sha512+(i*8)+7]  = state[i];
	}

	int difficulty = 10;
	
	verifyLeadingZeroes(hashed_array, difficulty, thread_offset_sha512, hashed_winner, idx);
}

__device__
void verifyLeadingZeroes(unsigned char *hash, int leading_zero, int hash_offset, bool* hashed_winner, int idx)
{
	for(int i=0;i<64;i++)
	{
		for (int j=0;j<8;j++)
		{
			if(leading_zero == 0)
			{
				hashed_winner[idx] = true;
				i = 64;
				break;
			}
			if(((hash[i+hash_offset] >> j) & 0x01) != 0)
			{
				i = 64;
				break;
			}
			else
				leading_zero--;
		}
	}
}

__global__
void padding(unsigned char *message, int size, unsigned char *hashed_array, 
	unsigned char *padded_array, bool* hashed_winner)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	//printf("X: %d\n", idx);

	int thread_offset = idx*128;
	int message_offset = idx*size;
	int pad_offset = 112;
	


	for(int i=0;i<size;i++)
		padded_array[i+thread_offset]=message[i+message_offset];
	
	padded_array[size+thread_offset] = 0x80;

	for(int i=size+1+thread_offset;i<pad_offset+thread_offset;i++)
		padded_array[i] = 0x00;

	uint64_t val = size*8;

	for(int i=pad_offset+thread_offset;i<pad_offset+thread_offset+8;i++)
		padded_array[i] = 0x00;

	padded_array[pad_offset+thread_offset+8]  =  val >> 56;
	padded_array[pad_offset+thread_offset+9]  =  val >> 48;
	padded_array[pad_offset+thread_offset+10] =  val >> 40;
	padded_array[pad_offset+thread_offset+11] =  val >> 32;
	padded_array[pad_offset+thread_offset+12] =  val >> 24;
    padded_array[pad_offset+thread_offset+13] =  val >> 16;
    padded_array[pad_offset+thread_offset+14] =  val >>  8;
    padded_array[pad_offset+thread_offset+15] =  val >>  0;

	computeHash((unsigned char*)padded_array, 128, (unsigned char*)hashed_array, idx, hashed_winner);
}

int main(void)
{
	int array_len = 1024;
	int cuda_blocks = 1;
	int string_len = 10;
	int counter = 0;

	clock_t t;
	t = clock();
	while(1)
	{
		unsigned char* temp_array = (unsigned char*)malloc(array_len*string_len*sizeof(unsigned char));

		unsigned char* message_array;
		unsigned char* hashed_array;
		unsigned char* padded_array;
		bool* hashed_winner;

		hipMallocManaged(&message_array, (cuda_blocks*array_len*string_len*sizeof(unsigned char)));
		hipMallocManaged(&hashed_array, (array_len*cuda_blocks*64*sizeof(unsigned char)));
		hipMallocManaged(&padded_array, (array_len*cuda_blocks*128*sizeof(unsigned char)));
		hipMallocManaged(&hashed_winner, (array_len*cuda_blocks*sizeof(bool)));
	
		temp_array = generateArray(array_len*cuda_blocks, string_len);
		for(int i=0; i<array_len*string_len;i++)
		{
			message_array[i] = temp_array[i];
		}

		free(temp_array);
		
		padding<<<cuda_blocks,array_len>>>(message_array, string_len, hashed_array, padded_array, hashed_winner);
		hipDeviceSynchronize();

	
		for(int i=0;i<array_len*cuda_blocks;i++)
		{
			if (hashed_winner[i] == true)
			{
			//	counter++;
				//printf("%d\n", counter);
				for(int j=0;j<64;j++)
					printf("%.2x", hashed_array[i*64+j]);
				printf("\n");
			}
		}
		

		hipFree(message_array);
		hipFree(hashed_array);
		hipFree(padded_array);
		hipFree(hashed_winner);
		//break;
	}
	t = clock() - t;
	double time_taken = t/CLOCKS_PER_SEC;
	//printf("%d hashes in %f seconds.\n", counter, time_taken);
	hipDeviceReset();
	return 0;
}